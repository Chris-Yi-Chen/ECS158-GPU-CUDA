#include "hip/hip_runtime.h"
// ./gaussian_blur_cuda city_256.pgm city_256_1.pgm 1
// nvcc -Xcompiler -Wall -Xcompiler -Werror ... -o gaussian_blur_cuda gaussian_blur_cuda.cu -lm

#include <stdio.h>
#include <stdlib.h>
#include <limits.h>
#include <math.h>

#define MAX_LINE_LENGTH 1024
#define M_PI 3.14159265358979323846

#define cuda_check(ret) _cuda_check((ret), __FILE__, __LINE__)
inline void _cuda_check(hipError_t ret, const char *file, int line)
{
	if (ret != hipSuccess) {
		fprintf(stderr, "CudaErr: %s (%s:%d)\n", hipGetErrorString(ret), file, line);
		exit(1);
	}
}


void gaussian_blur_cuda(int *img_h, int *outImg_h, float* kernel_h, int width, int height, int order) {

    dim3 grid_dim(1);
    dim3 block_dim(N, N);

}
/* 
1. Memory setup, data prep
2. Invoke the kernel
3. Retrieve results, memory cleanup
*/
void create_kernel_matrix(float **kernel, int order, float sigma) {
    *kernel = aligned_alloc(64, order * order * sizeof(float));
    int i, j;

    for (i = (int)(-order / 2); i <= (order / 2); i++) {
        for (j = (int)(-order / 2); j <= (order / 2); j++) {
            (*kernel)[(j + (order/2)) * order + (i + (order/2))] = gaussian_func(i, j, sigma);
        }
    }    
}

void write_pgm(char *filename, int* map, size_t N, int max_gray) {
    FILE* fp;
    size_t i;
    char* pixels;

    pixels = malloc(N * N);

    for (i = 0; i < N * N; i++) {
        pixels[i] = map[i];
    }

    /* Open file */
	fp = fopen(filename, "wb");
	if (!fp) {
		fprintf(stderr, "Error: cannot open file %s", filename);
		exit(1);
	}
    fprintf(fp, "P5\n%ld %ld\n%d\n", N, N, max_gray);
    fwrite(pixels, sizeof(char), N * N, fp);

    free(pixels);
    fclose(fp);

}

void read_pgm(const char *filename, int *width, int *height, int *max_gray, int **img) {
    int i;
    FILE *file = fopen(filename, "r");
    char line[MAX_LINE_LENGTH];
    // Read width and height, skipping comments
    while (fgets(line, sizeof(line), file)) {
        if (line[0] == '#') continue; // Skip comments

        if (sscanf(line, "%d %d", width, height) == 2) break;
    }

    while (fgets(line, sizeof(line), file)) {
        if (line[0] == '#') continue; // Skip comments

        if (sscanf(line, "%d", max_gray) == 1) break;
    }
    
    *img = aligned_alloc(64, (*width) * (*height) * sizeof(int));
    if (!(*img)) {
        perror("img alloc failed");
        fclose(file);
        exit(1);
    }

    for (i = 0; i < (*width) * (*height); i++) {
        int pixel = fgetc(file);
        if (pixel == EOF) {
            fprintf(stderr, "Unexpected end of file\n");
            free(*img);
            fclose(file);
            return;
        }
        (*img)[i] = pixel;
    }



    fclose(file);
}

void parse_float(char *str, char *val, float min, float max, float *num)
{
	float n = atof(str);
	if (n < min || n > max) {
		fprintf(stderr, "Error: wrong %s (%lf <= N <= %lf)", val, min, max);
		exit(1);
	}
	*num = n;
}

int main(int argc, char *argv[])
{
	float sigma;
    float *kernel;
    int width, height, max_gray;
    int *img;
    char *inFilename, *outFilename;

	/* Command line arguments */
	if (argc < 4) {
		fprintf(stderr, "Usage: %s <input_pgm> <output_pgm> <sigma>\n",
				argv[0]);
		exit(1);
	}

    inFilename = argv[1];
    read_pgm(inFilename, &width, &height, &max_gray, &img);
    outFilename = argv[2];
	parse_float(argv[3], "sigma",  0, 10, &sigma);

    /* Create Kernel Matrix */
    int order = (int)ceil(6.0 * sigma) % 2 == 1 ? ceil(6.0 * sigma) : ceil(6.0 * sigma) + 1; 
    create_kernel_matrix(&kernel, order, sigma);


    /* Call implementation */
    int *outImg = (int *)aligned_alloc(64, width * height * sizeof(int));
    gaussian_blur_cuda(img, outImg, kernel, width, height, order); 

    /* Save output image */
    write_pgm(outFilename, outImg, height, max_gray);

    /* Free resources */
    free(img);

    return 0;
}